#include "hip/hip_runtime.h"
/* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * */
/*   File:         cuda_kmeans.cu  (CUDA version)                            */
/*   Description:  Implementation of simple k-means clustering algorithm     */
/*                 This program takes an array of N data objects, each with  */
/*                 M coordinates and performs a k-means clustering given a   */
/*                 user-provided value of the number of clusters (K). The    */
/*                 clustering results are saved in 2 arrays:                 */
/*                 1. a returned array of size [K][N] indicating the center  */
/*                    coordinates of K clusters                              */
/*                 2. membership[N] stores the cluster center ids, each      */
/*                    corresponding to the cluster a data object is assigned */
/*                                                                           */
/*   Author:  Wei-keng Liao                                                  */
/*            ECE Department, Northwestern University                        */
/*            email: wkliao@ece.northwestern.edu                             */
/*   Copyright, 2005, Wei-keng Liao                                          */
/*                                                                           */
/* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * */

// Copyright (c) 2005 Wei-keng Liao
// Copyright (c) 2011 Serban Giuroiu
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
// THE SOFTWARE.

// -----------------------------------------------------------------------------

#include <stdio.h>
#include <stdlib.h>

#include "kmeans.h"
//#define PROFILING 0
#define MAX_THREADS (1024)

static inline int nextPowerOfTwo(int n) {
  n--;

  n = n >>  1 | n;
  n = n >>  2 | n;
  n = n >>  4 | n;
  n = n >>  8 | n;
  n = n >> 16 | n;
//  n = n >> 32 | n;    //  For 64-bit ints

  return ++n;
}

/*----< euclid_dist_2() >----------------------------------------------------*/
/* square of Euclid distance between two multi-dimensional points            */
__host__ __device__ inline static
float euclid_dist_2(int    numCoords,
                    int    numObjs,
                    int    numClusters,
                    float *objects,     // [numCoords][numObjs]
                    float *clusters,    // [numCoords][numClusters]
                    int    objectId,
                    int    clusterId)
{
  int i;
  float ans=0.0;

  for (i = 0; i < numCoords; i++) {
    ans += (objects[numObjs * i + objectId] - clusters[numClusters * i + clusterId]) *
        (objects[numObjs * i + objectId] - clusters[numClusters * i + clusterId]);
  }

  return(ans);
}

/*----< find_nearest_cluster() >---------------------------------------------*/
__global__ static
void find_nearest_cluster(int numCoords,
                          int numObjs,
                          int numClusters,
                          float *objects,           //  [numCoords][numObjs]
                          float *deviceClusters,    //  [numCoords][numClusters]
                          int *membership,          //  [numObjs]
                          int *intermediates)
{
  extern __shared__ char sharedMemory[];

  //  The type chosen for membershipChanged must be large enough to support
  //  reductions! There are blockDim.x elements, one for each thread in the
  //  block.
  unsigned char *membershipChanged = (unsigned char *)sharedMemory;
  float *clusters = (float *)(sharedMemory + blockDim.x);

  membershipChanged[threadIdx.x] = 0;

  //  BEWARE: We can overrun our shared memory here if there are too many
  //  clusters or too many coordinates!
  for (int i = threadIdx.x; i < numClusters; i += blockDim.x) {
    for (int j = 0; j < numCoords; j++) {
      clusters[numClusters * j + i] = deviceClusters[numClusters * j + i];
    }
  }
  __syncthreads();

  int objectId = blockDim.x * blockIdx.x + threadIdx.x;

  if (objectId < numObjs) {
    int index, i;
    float dist, min_dist;

    /* find the cluster id that has min distance to object */
    index = 0;
    min_dist = euclid_dist_2(numCoords, numObjs, numClusters,
                             objects, clusters, objectId, 0);

    for (i = 1; i < numClusters; i++) {
      dist = euclid_dist_2(numCoords, numObjs, numClusters,
                           objects, clusters, objectId, i);
      /* no need square root */
      if (dist < min_dist) { /* find the min and its array index */
        min_dist = dist;
        index = i;
      }
    }

    if (membership[objectId] != index) {
      membershipChanged[threadIdx.x] = 1;
    }

    /* assign the membership to object objectId */
    membership[objectId] = index;

    __syncthreads();    //  For membershipChanged[]

    //  blockDim.x *must* be a power of two!
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
      if (threadIdx.x < s) {
        membershipChanged[threadIdx.x] +=
            membershipChanged[threadIdx.x + s];
      }
      __syncthreads();
    }
  }
  if (threadIdx.x == 0) {
    intermediates[blockIdx.x] = membershipChanged[0];
  }
}

__global__ static
void compute_delta(int *deviceIntermediates,
                   int *deviceStore,
                   int numIntermediates,    //  The actual number of intermediates
                   int numIntermediates2)   //  The next power of two
{
  //  The number of elements in this array should be equal to
  //  numIntermediates2, the number of threads launched. It *must* be a power
  //  of two!
  extern __shared__ unsigned int intermediates[];

  int location = blockIdx.x * blockDim.x + threadIdx.x;

  //  Copy global intermediate values into shared memory.
  intermediates[threadIdx.x] = (location < numIntermediates) ? deviceIntermediates[location] : 0;
  __syncthreads();

  //  numIntermediates2 *must* be a power of two!
  for (unsigned int s = numIntermediates2 / 2; s > 0; s >>= 1) {
    if (threadIdx.x < s) {
      intermediates[threadIdx.x] += intermediates[threadIdx.x + s];
    }
    __syncthreads();
  }

  if (threadIdx.x == 0) {
    deviceStore[blockIdx.x] = intermediates[0];
  }
}


/**
 * This is a CUDA kernel which creates clusters and clusterArray using the
 * membership of every object. To reduce the contetion among threads from
 * various thread-blocks each thread block computes these datastructures in
 * their local shared memory and then finally copies them back to global
 * memory. Since, each thread-block creates a datastrutucre for its own
 * thread-block, they need to be reduced later by another kernel.
 * @param objects The pointer to device memory address holding the all
 * object's coordinate values.
 * @param deviceMembership The pointer to memory address holding the
 * membership information for all objects.
 * @param deviceComputeClusters The global memory where cluster array per
 * thread-block is stored.
 * @param deviceComputeClustersSize The global memory where clusterSize array
 * per thread-block is stored.
 * @param numObjects The actual number of objects
 * @param numClusters The number of clusters
 * @param numCoords The number of coordinates
 * @return
 */
__global__ static
void compute_membership(float *objects, // [numCoords][numObjs]
                    int *deviceMembership, // [numObjs]
                   float *deviceComputeClusters, //block_size][num_coords][num_clusters]
                   int *deviceComputeClustersSize, //[block_size][num_clusters]
                   int numObjects,
                   int numClusters,
                   int numCoords) {
  extern __shared__ int shared_memory[];
  int *clusterSize = shared_memory;
  float *clusters = (float*)(shared_memory + numClusters); // size of
                                            // clusterSize array is numClusters
  int threadCount = blockDim.x;
  int objectIndex = blockIdx.x * blockDim.x + threadIdx.x;
  int clusterIndex = blockIdx.x;

  // Let's not load the 0 values from global memory. It's slow. Initialize
  // the shared memory itself.

  int start = threadIdx.x;
  int maxPoints = numCoords * numClusters;

  // Initialize cluster size collectively
  if (threadIdx.x < numClusters) {
    clusterSize[threadIdx.x] = 0;
  }


  // numCoords * numClusters could be larger than threadCount
  while (start < maxPoints) {
    clusters[start] = 0.0;
    start += threadCount;
  }

  // Wait for load to complete
  __syncthreads();

  int newCluster = deviceMembership[objectIndex];
  if (objectIndex < numObjects) {
    // Reduce contention by having threads update different counters
    if (threadIdx.x < 512) {
      atomicAdd(&clusterSize[newCluster], 1);
      if (threadIdx.x > 256) {
        for (int i = 0; i < numCoords; i++) {
          atomicAdd(&clusters[i * numClusters + newCluster],
                    objects[i * numObjects + objectIndex]);
        }
      } else {
        for (int i = numCoords - 1; i >= 0; i--) {
          atomicAdd(&clusters[i * numClusters + newCluster],
                    objects[i * numObjects + objectIndex]);
        }
      }
    } else {
      if (threadIdx.x > 768) {
        for (int i = 0; i < numCoords; i++) {
          atomicAdd(&clusters[i * numClusters + newCluster],
                    objects[i * numObjects + objectIndex]);
        }
      } else {
        for (int i = numCoords - 1; i >= 0; i--) {
          atomicAdd(&clusters[i * numClusters + newCluster],
                    objects[i * numObjects + objectIndex]);
        }
      }
      atomicAdd(&clusterSize[newCluster], 1);
    }
  }

  // Wait for processing to complete
  __syncthreads();
  // Now store the data collectively to global memory at the block's clusterIndex
  if (threadIdx.x < numClusters) {
      int start = (clusterIndex * numClusters);
      deviceComputeClustersSize[start + threadIdx.x] = clusterSize[threadIdx.x];
  }

  int startIndex = clusterIndex * numClusters * numCoords;
  start = threadIdx.x;
  while (start < maxPoints) {
    deviceComputeClusters[startIndex + start] = clusters[start];
    start += threadCount;
  }
}

/**
 * This function retrieves the index given the clusterBlock, cooridnatorIndex
 * and clusterIndex.
 * @param numClusters The total number of clusters.
 * @param numCoord The total number of coordinates.
 * @param clusterBlock It's the index of thread-block from previous kernel.
 * @param coordIndex The coorindator Index
 * @param clusterIndex The clusterIndex
 * @return Returns the index in deviceComputeClusters array based on block
 * index, coordinator index and clusterindex.
 */
__device__ static inline int getIndex(int numClusters,
                                      int numCoord,
                                      int clusterBlock,
                                      int coordIndex,
                                      int clusterIndex) {
  int index = (clusterBlock * numClusters * numCoord) + (coordIndex * numClusters) + clusterIndex;
  return index;
}

/**
 * This function loads the coordinate value stored by previous kernel from
 * global memory based on the arguments provided. Each thread is trying to
 * reduce a different value hence this function will load value from
 * different index for every thread.
 * @param deviceComputeClusters The global memory where cluster array per
 * thread-block is stored.
 * @param numClusters The total number of clusters.
 * @param numCoord The total number of coordinates.
 * @param maxBlocks The maximum number of thread-block from previous kernel
 * we are trying to reduce.
 * @param clusterBlock It's the index of thread-block from previous kernel.
 * @param coordIndex The coorindator Index
 * @param clusterIndex The clusterIndex
 * @return It returns the coordinate value stored at the index.
 */
__device__ static inline float getIndexValue(float *deviceComputeClusters, //[prev_block_size][num_coords][num_clusters]
                                       int numClusters,
                                       int numCoord,
                                       int maxBlocks,
                                       int clusterBlock,
                                       int coordIndex,
                                       int clusterIndex) {
  if (clusterBlock < maxBlocks && clusterIndex < numClusters && coordIndex < numCoord) {
    int index = getIndex(numClusters, numCoord, clusterBlock, coordIndex, clusterIndex);
    return deviceComputeClusters[index];
  }
  return 0.0;
}

/**
 * This CUDA kernel reduces the cluster and clusterSize Array created by
 * compute-membership kernel at thread-block level. It finally copies the
 * reduced value back to global memory, which will later reduced by host
 * function. This kernel has grid dimension as (x, num_coords, num_clusters).
 * Each coordinator index and clusterIndex can be computed independently in a
 * different thread block thus providing a higher level of parallelism. Each
 * thread is responsible for a thread-block of previous kernel.
 * @param deviceComputeClusters The global memory where cluster array per
 * thread-block is stored.
 * @param deviceComputeClustersSize The global memory where clusterSize array
 * per thread-block is stored.
 * @param deviceClustersReduction The global memory, where the final
 * reduced result for clusters array will be stored.
 * @param deviceClustersSizeReduction The global memory, where the final
 * reduced result for clustersSize array will be stored.
 * @param numClusters The total number of clusters.
 * @param numCoord The total number of coordinates.
 * @param maxBlocks The maximum number of thread-block from previous kernel
 * we are trying to reduce.
 * @param numIntermediates2 The number of threads. It should always be a
 * power of 2 for binary reduction.
 * @return
 */
__global__ static
void reduce_clusters(float *deviceComputeClusters, //[prev_block_size][num_coords][num_clusters]
                    int * deviceComputeClustersSize, // [prev_block_size][num_clusters]
                   float *deviceClustersReduction, // [current_block_size][num_coords][num_clusters]
                   int *deviceClustersSizeReduction, // // [current_block_size][num_clusters]
                   int numClusters,
                   int numCoord,
                   int maxBlocks,
                   int numIntermediates2) {
  extern __shared__ float shared_memoryReduction[];
  float *intermediates = shared_memoryReduction;
  int *intermediateSize = (int*)(shared_memoryReduction + blockDim.x);

  int clusterBlock = blockIdx.x * blockDim.x + threadIdx.x;
  int coordIndex = blockIdx.y;
  int clusterIndex = blockIdx.z;
  //  Copy global intermediate values into shared memory.
  intermediates[threadIdx.x] = getIndexValue(deviceComputeClusters, numClusters, numCoord, maxBlocks, clusterBlock, coordIndex, clusterIndex);

  // If coodIndex is 0, also reduce the clusterSize array
  if (coordIndex == 0) {
    if (clusterBlock < maxBlocks)
    {
      intermediateSize[threadIdx.x] = deviceComputeClustersSize[clusterBlock * numClusters + clusterIndex];
    } else {
      intermediateSize[threadIdx.x] = 0;
    }
  }

  __syncthreads();
  //  numIntermediates2 *must* be a power of two!
  for (unsigned int s = numIntermediates2 / 2; s > 0; s >>= 1) {
    if (threadIdx.x < s) {
      intermediates[threadIdx.x] += intermediates[threadIdx.x + s];
      // Also reduce clusterSize
      if (coordIndex == 0) {
        intermediateSize[threadIdx.x] += intermediateSize[threadIdx.x + s];
      }
    }
    __syncthreads();
  }

  // One of the threads copies the reduced value to global memory
  if (threadIdx.x == 0) {
    deviceClustersReduction[blockIdx.x * numClusters * numCoord + coordIndex * numClusters + clusterIndex] = intermediates[0];
    // Also copy reduced size if reduced by this thread to global memory
    if (coordIndex == 0) {
      deviceClustersSizeReduction[blockIdx.x * numClusters + clusterIndex] = intermediateSize[0];
    }
  }
}

/*----< cuda_kmeans() >-------------------------------------------------------*/
//
//  ----------------------------------------
//  DATA LAYOUT
//
//  objects         [numObjs][numCoords]
//  clusters        [numClusters][numCoords]
//  dimObjects      [numCoords][numObjs]
//  dimClusters     [numCoords][numClusters]
//  newClusters     [numCoords][numClusters]
//  deviceObjects   [numCoords][numObjs]
//  deviceClusters  [numCoords][numClusters]
//  ----------------------------------------
//
/* return an array of cluster centers of size [numClusters][numCoords]       */
float** cuda_kmeans(float **objects,      /* in: [numObjs][numCoords] */
                    int     numCoords,    /* no. features */
                    int     numObjs,      /* no. objects */
                    int     numClusters,  /* no. clusters */
                    float   threshold,    /* % objects change membership */
                    int    *membership,   /* out: [numObjs] */
                    int    *loop_iterations)
{
  int      i, j, loop=0;
  int     *newClusterSize; /* [numClusters]: no. objects assigned in each
                                new cluster */
  float    delta;          /* % of objects change their clusters */
  float  **dimObjects;
  float  **clusters;       /* out: [numClusters][numCoords] */
  float  **dimClusters;
  float  **newClusters;    /* [numCoords][numClusters] */

  float *deviceObjects;
  float *deviceClusters;
  int *deviceMembership;
  int *deviceIntermediates;

  float *deviceComputeClusters;
  int *deviceComputeClustersSize;

  int *deviceDeltaReduction;
  int *hostDeltaReduction;
  int *deviceClusterSizeReduction;
  int *hostClusterSizeReduction;
  float *deviceClustersReduction;
  float *hostClustersReduction;

  //  Copy objects given in [numObjs][numCoords] layout to new
  //  [numCoords][numObjs] layout
  malloc2D(dimObjects, numCoords, numObjs, float);
  for (i = 0; i < numCoords; i++) {
    for (j = 0; j < numObjs; j++) {
      dimObjects[i][j] = objects[j][i];
    }
  }

  /* pick first numClusters elements of objects[] as initial cluster centers*/
  malloc2D(dimClusters, numCoords, numClusters, float);
  for (i = 0; i < numCoords; i++) {
    for (j = 0; j < numClusters; j++) {
      dimClusters[i][j] = dimObjects[i][j];
    }
  }

  /* initialize membership[] */
  for (i=0; i<numObjs; i++) membership[i] = -1;

  /* need to initialize newClusterSize and newClusters[0] to all 0 */
  newClusterSize = (int*) calloc(numClusters, sizeof(int));
  assert(newClusterSize != NULL);

  malloc2D(newClusters, numCoords, numClusters, float);
  memset(newClusters[0], 0, numCoords * numClusters * sizeof(float));

  //  To support reduction, numThreadsPerClusterBlock *must* be a power of
  //  two, and it *must* be no larger than the number of bits that will
  //  fit into an unsigned char, the type used to keep track of membership
  //  changes in the kernel.
  const unsigned int numThreadsPerClusterBlock = 128;
  const unsigned int numClusterBlocks =
      (numObjs + numThreadsPerClusterBlock - 1) / numThreadsPerClusterBlock;
  const unsigned int clusterBlockSharedDataSize =
      numThreadsPerClusterBlock * sizeof(unsigned char) +
          numClusters * numCoords * sizeof(float);

  // Below datastructures are initialized for Apoorv's implementation

  /* Below variables are used for delta reduction kernel*/

  /* It should be clusterBlocks divided by number of threads*/
  int deltaReductionBlocks = (numClusterBlocks + MAX_THREADS - 1) / MAX_THREADS;

  /* Total number of threads should be power of 2 for binary reduction */
  int deltaReductionThreads = min(MAX_THREADS, numClusterBlocks);
  int deltaReductionPow2 = nextPowerOfTwo(deltaReductionThreads);

  /* Each thread needs a shared memory to load the delta value */
  const unsigned int deltaReductionSharedDataSize = MAX_THREADS * sizeof(unsigned int);

  /* Allocate memory so that each thread-block could store their final values
   * after reduction to a different location. The host code will finally
   * aggregate them*/
  hostDeltaReduction = (int *)malloc(sizeof(int) * deltaReductionBlocks);
  assert(hostDeltaReduction != NULL);
  checkCuda(hipMalloc(&deviceDeltaReduction, deltaReductionBlocks*sizeof(int)));

  /* Below variables are used for computing clusters using membership kernel*/
  const unsigned int computeClusterObjectBlocks = (numObjs + MAX_THREADS - 1) / MAX_THREADS;
  /* Each thread block requires to store clusters and clusterSize in their
   * shared memory */
  const unsigned int computeClusterSharedDataSize = sizeof(int) * numClusters + sizeof(float) * numClusters * numCoords;

  /*Allocate enough memory for all thread-blocks */
  checkCuda(hipMalloc(&deviceComputeClusters, computeClusterObjectBlocks*numClusters*numCoords*sizeof(float)));
  checkCuda(hipMalloc(&deviceComputeClustersSize, computeClusterObjectBlocks*numClusters*sizeof(int)));

  /* Below variables are used to reduce the clusters and clusterSize from
   * previous kernel */

  /* We have total computeClusterObjectBlocks different datastructures. Let
   * one thread handle each of them */
  int clusterReductionBlocks = (computeClusterObjectBlocks + MAX_THREADS - 1) / MAX_THREADS;
  /* For binary reduction, it is necessary to have power of two as number of
   * threads */
  int clusterReductionThreads = min(MAX_THREADS, computeClusterObjectBlocks);
  int clusterReductionPow2 = nextPowerOfTwo(clusterReductionThreads);

  /* Each thread-block is reducing the size and coordinate value at the same
   * time. Hence we need shared memory for both of them */
  const unsigned int clusterReductionSharedDataSize = sizeof(float) *
      MAX_THREADS + sizeof(int) * MAX_THREADS;

  /* Every coordinate and cluster is independent of each other and can be
   * executed in parallel. We will use X axis as block index, Y axis as
   * coordinator index and z-axis as clusterIndex */
  dim3 blockDim(clusterReductionBlocks, numCoords, numClusters);

  /*Allocate enough host and device memory so that this kernel can store the
   * reduced data in global memory. This data is copied to host where it
   * calculate the final values of clusters and clusterSize */
  hostClusterSizeReduction = (int*)malloc(sizeof(int) *
      clusterReductionBlocks * numClusters);
  assert(hostClusterSizeReduction != NULL);
  hostClustersReduction = (float*)malloc(sizeof(float) *
      clusterReductionBlocks * numClusters * numCoords);
  assert(hostClustersReduction != NULL);
  checkCuda(hipMalloc(&deviceClusterSizeReduction, clusterReductionBlocks
      * numClusters * sizeof(int)));
  checkCuda(hipMalloc(&deviceClustersReduction, clusterReductionBlocks
      * numCoords * numClusters * sizeof(float)));

  /**Optimization related datastructures ready here*/

  checkCuda(hipMalloc(&deviceObjects, numObjs*numCoords*sizeof(float)));
  checkCuda(hipMalloc(&deviceClusters, numClusters*numCoords*sizeof(float)));
  checkCuda(hipMalloc(&deviceMembership, numObjs*sizeof(int)));
  checkCuda(hipMalloc(&deviceIntermediates, deltaReductionThreads*sizeof(unsigned int)));

  /* Copy initial objects and membership to device memory */
  checkCuda(hipMemcpy(deviceObjects, dimObjects[0],
                       numObjs*numCoords*sizeof(float), hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(deviceMembership, membership,
                       numObjs*sizeof(int), hipMemcpyHostToDevice));

/* The code inside #ifdef PROFILING will be compiled out when PROFILING is
 * not defined. If PROFILING is required please pass a flag as -dPROFILING as
 * compile time parameters. This PROFILING code is really helpful to profile
 * various sections of the implementation */
#ifdef PROFILING
  double findTime = 0;
  double computeTime = 0;
  double newClusterTime = 0;
  double newReductionTime = 0;
  double totalTime = wtime();
#endif
  do {
    /* First copy the clusters back to device memory */
    checkCuda(hipMemcpy(deviceClusters, dimClusters[0],
                         numClusters*numCoords*sizeof(float), hipMemcpyHostToDevice));
#ifdef PROFILING
    double beforeFind = wtime();
#endif
    /* Calculate new membership for every object using eucledian method */
    find_nearest_cluster
        <<< numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize >>>
        (numCoords, numObjs, numClusters,
            deviceObjects, deviceClusters, deviceMembership, deviceIntermediates);

//    hipDeviceSynchronize(); checkLastCudaError();
#ifdef PROFILING
    double afterFind = wtime();
#endif
    /* This kernel reduces the delta calculated by every thread block in
     * previous kernel */
    compute_delta <<< deltaReductionBlocks, deltaReductionPow2, deltaReductionSharedDataSize >>>
            (deviceIntermediates, deviceDeltaReduction, numClusterBlocks, deltaReductionPow2);
//    hipDeviceSynchronize(); checkLastCudaError();
    // Now aggregate the delta after copying from device
    checkCuda(hipMemcpy(hostDeltaReduction, deviceDeltaReduction,
                         sizeof(int) * deltaReductionBlocks, hipMemcpyDeviceToHost));
    int d = 0;
    for (int i = 0; i < deltaReductionBlocks; i++) {
      d += hostDeltaReduction[i];
    }
#ifdef PROFILING
    double afterCompute = wtime();
#endif
    /* This kernel uses the membership array to calculate new cluster and
     * clusterSize array per thread block. This needs to be reduced later */
    compute_membership <<< computeClusterObjectBlocks, MAX_THREADS, computeClusterSharedDataSize>>>
            (   deviceObjects,
                deviceMembership,
                deviceComputeClusters,
                deviceComputeClustersSize,
                numObjs,    //  The actual number of objects
                numClusters,  //  The number of clusters
                numCoords);
//    hipDeviceSynchronize(); checkLastCudaError();
#ifdef PROFILING
    double kTime = wtime();
#endif
    /* Reduce the clusters and clustersSize array created per thread-block in
     * previous kernel function */
    reduce_clusters <<< blockDim, clusterReductionPow2,
        clusterReductionSharedDataSize >>> (deviceComputeClusters,
            deviceComputeClustersSize, deviceClustersReduction,
            deviceClusterSizeReduction, numClusters,
            numCoords, computeClusterObjectBlocks, clusterReductionPow2);

    // Now aggregate the clusters and clusterSize after copying from device
    checkCuda(hipMemcpy(hostClusterSizeReduction, deviceClusterSizeReduction,
                         sizeof(int) * clusterReductionBlocks * numClusters, hipMemcpyDeviceToHost));
    checkCuda(hipMemcpy(hostClustersReduction, deviceClustersReduction,
                         sizeof(float) * clusterReductionBlocks * numCoords * numClusters, hipMemcpyDeviceToHost));
    // Agrregate the clustersSize
    for (int i = 0; i < numClusters; i++) {
      newClusterSize[i] = 0;
      for (int j = 0; j < clusterReductionBlocks; j++) {
        newClusterSize[i] += hostClusterSizeReduction[j * numClusters + i];
      }
    }

    // Aggregate the clusters. This is really quick code. PROFILING clearly
    // indicates that this was never a bottleneck in our implementation
    for (int i = 0; i < numClusters; i++) {
      for (int j = 0; j < numCoords; j++) {
        dimClusters[j][i] = 0.0;
        for (int k = 0; k < clusterReductionBlocks; k++) {
          dimClusters[j][i] += hostClustersReduction[k * numClusters * numCoords + j * numClusters + i];
        }
        // Average based on it's size
        if (newClusterSize[i] != 0) {
          dimClusters[j][i] /= newClusterSize[i];
        } else {
          dimClusters[j][i] = 0.0;
        }
      }
    }
#ifdef PROFILING
    double afterReduction = wtime();
    findTime += (afterFind - beforeFind);
    computeTime += (afterCompute - afterFind);
    newClusterTime += (kTime - afterCompute);
    newReductionTime += (afterReduction - afterCompute);
#endif
    delta = d/(float)numObjs;
  } while (delta > threshold && loop++ < 500);

#ifdef PROFILING
  totalTime = wtime() - totalTime;
  printf("Total Time: %10.4f\n", totalTime);
  printf("Total Find Time: %10.4f\n", findTime);
  printf("Total Compute Time: %10.4f\n", computeTime);
  printf("Total New cluster Time: %10.4f\n", newClusterTime);
  printf("Total New Reduction Time: %10.4f\n", newReductionTime);
#endif
  *loop_iterations = loop + 1;

  checkCuda(hipMemcpy(membership, deviceMembership,
                         numObjs*sizeof(int), hipMemcpyDeviceToHost));

  /* allocate a 2D space for returning variable clusters[] (coordinates
     of cluster centers) */
  malloc2D(clusters, numClusters, numCoords, float);
  for (i = 0; i < numClusters; i++) {
    for (j = 0; j < numCoords; j++) {
      clusters[i][j] = dimClusters[j][i];
    }
  }

  checkCuda(hipFree(deviceObjects));
  checkCuda(hipFree(deviceClusters));
  checkCuda(hipFree(deviceMembership));
  checkCuda(hipFree(deviceIntermediates));
  checkCuda(hipFree(deviceComputeClusters));
  checkCuda(hipFree(deviceComputeClustersSize));
  checkCuda(hipFree(deviceClusterSizeReduction));
  checkCuda(hipFree(deviceClustersReduction));
  checkCuda(hipFree(deviceDeltaReduction));

  free(dimObjects[0]);
  free(dimObjects);
  free(dimClusters[0]);
  free(dimClusters);
  free(newClusters[0]);
  free(newClusters);
  free(newClusterSize);
  free(hostClusterSizeReduction);
  free(hostClustersReduction);
  free(hostDeltaReduction);

  return clusters;
}